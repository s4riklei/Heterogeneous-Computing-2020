
#include "hip/hip_runtime.h"


#include <iostream>
#include <stdexcept>
#include <string>
#include <random>
#include <vector>
#include <utility>
#include <cstddef>
#include <cfloat>
#include <chrono>

namespace {

	struct Metadata {
		int desiredBlockCount;		// minimum number of blocks to create
		int threadCountPerBlock;	// threads per block
		int pointCount;				// number of points for the traveling salesman to visit
		int randomSeed;				// random seed used to generate the distance matrix
		int threadCount;			// actual number of generated threads
		int prefixSize;				// amount of static points per thread, that don't get permutated
	};

	struct DevicePointers {
		float* distanceMatrix = 0;
		// memory block containing the best result permutations found by each thread (device)
		int* allThreadsBestResultPermutations = 0;
		// array containing the best distance found by each thread (device)
		float* allThreadsBestResults = 0;
		// memory block containing the current permutation state threads are operating on
		int* allThreadsPermutations = 0;
		// array containing the temp state while executing Heap's algorithm
		int* allThreadsC = 0;
	};

	Metadata parseArgs(int argc, char** argv) {
		int desiredBlockCount;
		int threadCountPerBlock;
		int pointCount;
		int randomSeed;
		int threadCount;
		int prefixSize;

		// validate user input
		if (argc != 5) {
			throw std::invalid_argument("Error: Incorrect number of arguments");
		}

		try {
			desiredBlockCount = std::stoi(argv[1]);
			if (desiredBlockCount < 1) {
				throw std::invalid_argument("Error: Please set block count to a value greater than 0");
			}

			threadCountPerBlock = std::stoi(argv[2]);
			if (threadCountPerBlock < 1 || threadCountPerBlock > 1024) {
				throw std::invalid_argument("Error: Please set thread count per block to a value between 1 and 1024");
			}

			pointCount = std::stoi(argv[3]);
			if (pointCount < 2) {
				throw std::invalid_argument("Error: Please set point count to a value greater than 1");
			}

			randomSeed = std::stoi(argv[4]);
		}
		catch (const std::out_of_range& e) {
			std::cout << e.what() << std::endl;
			throw std::invalid_argument("Error: A parameter value was out of range");
		}

		// calculate thread count and prefix size
		([&] {
			int desiredThreadCount = desiredBlockCount * threadCountPerBlock;
			threadCount = 1;
			prefixSize = 0;
			for (int i = pointCount; (i > 0 && threadCount < desiredThreadCount); i--) {
				threadCount *= i;
				prefixSize++;
			}
		})();

		Metadata m{
			desiredBlockCount,
			threadCountPerBlock,
			pointCount,
			randomSeed,
			threadCount,
			prefixSize
		};
		return m;
	}

	std::vector<std::vector<float>> createDistanceMatrix(const Metadata& metadata) {
		std::default_random_engine generator(metadata.randomSeed);
		std::uniform_real_distribution<float> distribution(0.0f, 1000.0f);

		std::vector<std::vector<float>> matrix;
		matrix.reserve(metadata.pointCount);
		for (int i = 0; i < metadata.pointCount; i++) {
			std::vector<float> row;
			row.reserve(metadata.pointCount);
			for (int i = 0; i < metadata.pointCount; i++) {
				row.push_back(distribution(generator));
			}
			matrix.push_back(std::move(row));
		}

		return matrix;
	}

	void printDistanceMatrix(const std::vector<std::vector<float>>& matrix) {
		std::cout << "Distance Matrix:" << std::endl;
		std::cout << "------" << "\t";
		for (int i = 0; i < matrix.size(); i++) {
			std::cout << i << ":\t";
		}
		std::cout << std::endl;

		int counter = 0;
		for (auto& row : matrix) {
			std::cout << counter++ << ":\t";
			for (auto& val : row) {
				std::cout << val << "\t";
			}
			std::cout << std::endl;
		}
	}

	DevicePointers allocateDeviceMemory(const Metadata& metadata) {
		hipError_t cudaStatus;

		DevicePointers d;

		// since metadata.threadCount is probably not always divisible by metadata.threadCountPerBlock,
		// some "overhead" memory needs to be allocated to not cause segfaults when coalescing memory
		int actualThreadCount = ([&] {
			int result = metadata.threadCount / metadata.threadCountPerBlock;
			result += metadata.threadCount % metadata.threadCountPerBlock == 0 ? 0 : 1;
			result *= metadata.threadCountPerBlock;
			return result;
		})();

		std::size_t size = metadata.pointCount * metadata.pointCount * sizeof(float);
		cudaStatus = hipMalloc(&d.distanceMatrix, size);
		if (cudaStatus != hipSuccess) {
			throw std::runtime_error("Error allocating memory!");
		}

		size = actualThreadCount * metadata.pointCount * sizeof(int);
		cudaStatus = hipMalloc(&d.allThreadsBestResultPermutations, size);
		if (cudaStatus != hipSuccess) {
			throw std::runtime_error("Error allocating memory!");
		}

		size = actualThreadCount * sizeof(float);
		cudaStatus = hipMalloc(&d.allThreadsBestResults, size);
		if (cudaStatus != hipSuccess) {
			throw std::runtime_error("Error allocating memory!");
		}

		size = actualThreadCount * metadata.pointCount * sizeof(int);
		cudaStatus = hipMalloc(&d.allThreadsPermutations, size);
		if (cudaStatus != hipSuccess) {
			throw std::runtime_error("Error allocating memory!");
		}

		size = actualThreadCount * (metadata.pointCount - metadata.prefixSize) * sizeof(int);
		cudaStatus = hipMalloc(&d.allThreadsC, size);
		if (cudaStatus != hipSuccess) {
			throw std::runtime_error("Error allocating memory!");
		}

		return d;
	}

	void freeDeviceMemory(DevicePointers& d) {
		hipFree(d.distanceMatrix);
		hipFree(d.allThreadsBestResultPermutations);
		hipFree(d.allThreadsBestResults);
		hipFree(d.allThreadsPermutations);
		hipFree(d.allThreadsC);

		d.distanceMatrix = 0;
		d.allThreadsBestResultPermutations = 0;
		d.allThreadsBestResults = 0;
		d.allThreadsPermutations = 0;
		d.allThreadsC = 0;
	}

	void copyDataHostToDevice(const DevicePointers& d, const std::vector<std::vector<float>>& distanceMatrix) {
		float* currentPointer = d.distanceMatrix;
		hipError_t cudaStatus;
		for (auto& row : distanceMatrix) {
			std::size_t count = row.size() * sizeof(float);
			cudaStatus = hipMemcpy(currentPointer, row.data(), count, hipMemcpyHostToDevice);
			if (cudaStatus != hipSuccess) {
				throw std::runtime_error("Error copying distance matrix onto device!");
			}
			currentPointer += row.size();
		}
	}

	__device__ float calculatePermutationTotalDistance(float* distanceMatrix, int* permutation, int pointCount) {
		float distance = distanceMatrix[permutation[blockDim.x * (pointCount - 1)] * pointCount + permutation[0]];
		for (int i = 0; i < pointCount - 1; i++) {
			distance += distanceMatrix[permutation[blockDim.x * i] * pointCount + permutation[blockDim.x * (i + 1)]];
		}
		return distance;
	}

	__device__ void swap(int& a, int& b) {
		int tmp = a;
		a = b;
		b = tmp;
	}

	__device__ void copyPermutationToBestArray(int* dest, int* src, int size) {
		for (int i = 0; i < size; i++) {
			dest[i] = src[blockDim.x * i];
		}
	}

	__device__ void prepareSharedDistanceMatrix(const int id, const int pointCount, const float* global, float* shared) {
		int size = pointCount * pointCount;
		for (int i = threadIdx.x; i < size; i += blockDim.x) {
			shared[i] = global[i];
		}
	}

	__global__ void calculateBestRoute(const int threadCount, const int prefixSize, const int pointCount, const DevicePointers d) {
		int id = blockIdx.x * blockDim.x + threadIdx.x;
		if (threadCount <= id) {
			return;
		}
		
		// initialize permutation
		int* permutation = d.allThreadsPermutations + blockIdx.x * blockDim.x * pointCount + threadIdx.x;

		for (int i = 0; i < pointCount; i++) {
			permutation[blockDim.x * i] = i;
		}

		int quotient = id;
		for (int i = prefixSize - 1; i >= 0; i--) {
			int factoradic = quotient % (pointCount - i);
			quotient /= pointCount - i;
			swap(permutation[blockDim.x * i], permutation[blockDim.x * (factoradic + i)]);
		}

		// prepare pointers
		int* permutable = permutation +  blockDim.x * prefixSize;
		int permutableSize = pointCount - prefixSize;
		int* bestResultPermutations = d.allThreadsBestResultPermutations + id * pointCount;
		extern __shared__ float sharedDistanceMatrix[];

		prepareSharedDistanceMatrix(id, pointCount, d.distanceMatrix, sharedDistanceMatrix);
		__syncthreads();

		// initial solution is initial best
		copyPermutationToBestArray(bestResultPermutations, permutation, pointCount);
		float bestResult = calculatePermutationTotalDistance(sharedDistanceMatrix, permutation, pointCount);

		if (permutableSize != 0) {
			int* c = d.allThreadsC + blockIdx.x * blockDim.x * permutableSize + threadIdx.x;
			for (int i = 0; i < permutableSize; i++) {
				c[blockDim.x * i] = 0;
			}
			int i = 0;
			while (i < permutableSize) {
				if (c[blockDim.x * i] < i) {
					if (i % 2 == 0) {
						swap(permutable[0], permutable[blockDim.x * i]);
					}
					else {
						swap(permutable[blockDim.x * c[blockDim.x * i]], permutable[blockDim.x * i]);
					}

					float tempResult = calculatePermutationTotalDistance(sharedDistanceMatrix, permutation, pointCount);
					if (tempResult < bestResult) {
						copyPermutationToBestArray(bestResultPermutations, permutation, pointCount);
						bestResult = tempResult;
					}


					c[blockDim.x * i]++;
					i = 0;
				}
				else {
					c[blockDim.x * i] = 0;
					i++;
				}
			}
		}

		d.allThreadsBestResults[id] = bestResult;
	}

	void executeThreads(const Metadata& metadata, const DevicePointers& d) {
		int blockCount = ([&] {
			int result = metadata.threadCount / metadata.threadCountPerBlock;
			result += metadata.threadCount % metadata.threadCountPerBlock == 0 ? 0 : 1;
			return result;
		})();
		std::size_t sharedMemorySize = metadata.pointCount * metadata.pointCount * sizeof(float);
		calculateBestRoute<<<blockCount, metadata.threadCountPerBlock, sharedMemorySize>>>(metadata.threadCount, metadata.prefixSize, metadata.pointCount, d);
	}

	void copyDataDeviceToHostAndEvaluate(const Metadata& metadata, const DevicePointers& d) {
		// copy data
		hipError_t cudaStatus;
		
		std::vector<std::vector<int>> allThreadsBestResultPermutations = ([&] {
			std::vector<std::vector<int>> result;
			result.reserve(metadata.threadCount);
			int* tmp = new int[metadata.threadCount * metadata.pointCount];
			std::size_t size = metadata.threadCount * metadata.pointCount * sizeof(int);
			cudaStatus = hipMemcpy(tmp, d.allThreadsBestResultPermutations, size, hipMemcpyDeviceToHost);
			if (cudaStatus != hipSuccess) {
				throw std::runtime_error("Error copying data from device to host");
			}

			int* intervalStart = tmp;
			int* intervalEnd = tmp + metadata.pointCount;
			for (int i = 0; i < metadata.threadCount; i++) {
				result.push_back(std::vector<int>(intervalStart, intervalEnd));
				intervalStart += metadata.pointCount;
				intervalEnd += metadata.pointCount;
			}

			delete[] tmp;
			return result;
		})();

		std::vector<float> allThreadsBestResults = ([&] {
			float* tmp = new float[metadata.threadCount];
			std::size_t size = metadata.threadCount * sizeof(float);
			cudaStatus = hipMemcpy(tmp, d.allThreadsBestResults, size, hipMemcpyDeviceToHost);
			if (cudaStatus != hipSuccess) {
				throw std::runtime_error("Error copying data from device to host");
			}

			std::vector<float> result(tmp, tmp + metadata.threadCount);
			delete[] tmp;
			return result;
		})();

		// evaluate
		float bestResult = FLT_MAX;
		std::vector<int>* bestResultPermutation = 0;
		for (int i = 0; i < allThreadsBestResults.size(); i++) {
			if (allThreadsBestResults[i] < bestResult) {
				bestResult = allThreadsBestResults[i];
				bestResultPermutation = &allThreadsBestResultPermutations[i];
			}
		}

		std::cout << "Best Route:" << std::endl;
		for (int i = 0; i < bestResultPermutation->size(); i++) {
			std::cout << bestResultPermutation->at(i) << " -> ";
		}
		std::cout << bestResultPermutation->at(0) << std::endl;
		std::cout << "Route Length: " << bestResult << std::endl;
	}
}

int main(int argc, char** argv) {
	DevicePointers devicePointers;

	try {
		std::chrono::steady_clock::time_point t_start = std::chrono::steady_clock::now();

		const Metadata METADATA = parseArgs(argc, argv);
		
		std::vector<std::vector<float>> distanceMatrix = createDistanceMatrix(METADATA);
		printDistanceMatrix(distanceMatrix);

		devicePointers = allocateDeviceMemory(METADATA);

		copyDataHostToDevice(devicePointers, distanceMatrix);

		std::chrono::steady_clock::time_point t_threadStart = std::chrono::steady_clock::now();
		executeThreads(METADATA, devicePointers);
		hipDeviceSynchronize();
		std::chrono::steady_clock::time_point t_threadEnd = std::chrono::steady_clock::now();

		copyDataDeviceToHostAndEvaluate(METADATA, devicePointers);

		freeDeviceMemory(devicePointers);

		hipDeviceReset();

		std::chrono::steady_clock::time_point t_end = std::chrono::steady_clock::now();
		std::chrono::duration<double> totalExecutionTime = std::chrono::duration_cast<std::chrono::duration<double>>(t_end - t_start);
		std::chrono::duration<double> threadExecutionTime = std::chrono::duration_cast<std::chrono::duration<double>>(t_threadEnd - t_threadStart);
		std::cout << "Total execution time (s): " << totalExecutionTime.count() << std::endl;
		std::cout << "Thread execution time (s): " << threadExecutionTime.count() << std::endl;
	}
	catch (const std::invalid_argument& e) {
		std::cerr << e.what() << std::endl;
		std::cerr << "Usage: " << argv[0] << "<desired block count (int)> <thread count per block (int)>" <<
			" <point count (int)> <random seed (int)>" << std::endl;
		std::cerr << "desired block count = Minimum number of blocks to create" << std::endl;
		std::cerr << "thread count per block = Number of threads for each block" << std::endl;
		std::cerr << "point count = Number of points to be generated for the traveling salesman problem" << std::endl;
		std::cerr << "random seed = random seed used to generate the distances between points" << std::endl;
		return 1;
	}
	catch (const std::runtime_error& e) {
		std::cerr << e.what() << std::endl;
		freeDeviceMemory(devicePointers);
		return 1;
	}
}